#include "hip/hip_runtime.h"
/*
 * marching_cube.cu
 *
 *  Created on: Sep 7, 2012
 *      Author: ollie
 */

#include <sys/time.h>

#include <vtkImageData.h>
#include <vtkRTAnalyticSource.h>

#include <thrust/extrema.h>

#include "piston/marching_cube.h"
#include "piston/util/tangle_field.h"

#include "Stats.h"

static const float ISO_START=16;
static const float ISO_STEP=-1;
static const int ISO_NUM=15;
static const size_t DIMENSIONS = 128;

//#define SPACE thrust::host_space_tag
#define SPACE thrust::device_space_tag

using namespace piston;

int main()
{
  const int grid_size = DIMENSIONS + 1;

  tangle_field<SPACE>* tangle;
  marching_cube<tangle_field<SPACE>, tangle_field<SPACE> > *isosurface;

  tangle = new tangle_field<SPACE>(grid_size, grid_size, grid_size);
  isosurface = new marching_cube<tangle_field<SPACE>,  tangle_field<SPACE> >(*tangle, *tangle, ISO_START);

  std::vector<double> samples;

  const double MAX_RUNTIME = 30;
  const size_t MAX_ITERATIONS = 1;
  samples.reserve(MAX_ITERATIONS);

  size_t iter = 0;
  Timer timer;
  for (double el = 0.0; el < MAX_RUNTIME && iter < MAX_ITERATIONS; el += samples.back(), ++iter)
  {
    float isovalue = ISO_START;
    timer.Reset();
    for (int j=0; j<ISO_NUM; j++)
    {
      isovalue += ISO_STEP;
      isosurface->set_isovalue(isovalue);
      (*isosurface)();
      std::cout << isovalue << " " << isosurface->num_total_vertices << std::endl;
    }
    samples.push_back(timer.GetElapsedTime());
  }

  std::sort(samples.begin(), samples.end());
  stats::Winsorize(samples, 5.0);
  std::cout << "Benchmark \'VTK MPI Isosurface\' results:\n"
      << "\tmedian = " << stats::PercentileValue(samples, 50.0) << "s\n"
      << "\tmedian abs dev = " << stats::MedianAbsDeviation(samples) << "s\n"
      << "\tmean = " << stats::Mean(samples) << "s\n"
      << "\tstd dev = " << stats::StandardDeviation(samples) << "s\n"
      << "\tmin = " << samples.front() << "s\n"
      << "\tmax = " << samples.back() << "s\n"
      << "\t# of runs = " << samples.size() << "\n";


    return 0;
}



